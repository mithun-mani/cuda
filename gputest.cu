/*#include <stdio.h>
int main() {
    printf("Hello World from GPU!\n");
}

__global__
void printer (){
    printf("Hello World from GPU\n");
}
int main() {
    printer<<<1,1>>>();
    return 0;
}
*/

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_from_gpu(void) {
    printf("Hello World! from thread [%d,%d] \
        From device\n", threadIdx.x,blockIdx.x);
}

int main(void) {
    printf("Hello World from host!\n");
    print_from_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
